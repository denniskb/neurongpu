#include "hip/hip_runtime.h"
/*
Copyright (C) 2020 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <config.h>
#include <stdio.h>
#include <stdlib.h>

				    //#include "cuda_error_nl.h"
#include "cuda_error.h"
#include "nested_loop.h"

//TMP
#include "getRealTime.h"
//

//////////////////////////////////////////////////////////////////////
// declare here the functions called by the nested loop 
__device__ void NestedLoopFunction0(int ix, int iy);
__device__ void NestedLoopFunction1(int ix, int iy);
//////////////////////////////////////////////////////////////////////

namespace NestedLoop
{
  PrefixScan prefix_scan_;
  int *d_Ny_cumul_sum_;   
}

__device__ int locate(int val, int *data, int n)
{
  int i_left = 0;
  int i_right = n-1;
  int i = (i_left+i_right)/2;
  while(i_right-i_left>1) {
    if (data[i] > val) i_right = i;
    else if (data[i]<val) i_left = i;
    else break;
    i=(i_left+i_right)/2;
  }

  return i;
}

__global__ void CumulSumNestedLoopKernel0(int Nx, int *Ny_cumul_sum,
					 int Ny_sum)
{
  int blockId   = blockIdx.y * gridDim.x + blockIdx.x;
  int array_idx = blockId * blockDim.x + threadIdx.x;
  if (array_idx<Ny_sum) {
    int ix = locate(array_idx, Ny_cumul_sum, Nx + 1);
    int iy = (int)(array_idx - Ny_cumul_sum[ix]);
    NestedLoopFunction0(ix, iy);
  }
}

__global__ void CumulSumNestedLoopKernel1(int Nx, int *Ny_cumul_sum,
					 int Ny_sum)
{
  int blockId   = blockIdx.y * gridDim.x + blockIdx.x;
  int array_idx = blockId * blockDim.x + threadIdx.x;
  if (array_idx<Ny_sum) {
    int ix = locate(array_idx, Ny_cumul_sum, Nx + 1);
    int iy = (int)(array_idx - Ny_cumul_sum[ix]);
    NestedLoopFunction1(ix, iy);
  }
}

//////////////////////////////////////////////////////////////////////
int NestedLoop::Init()
{
  //prefix_scan_.Init();
  gpuErrchk(hipMalloc(&d_Ny_cumul_sum_,
			  PrefixScan::AllocSize*sizeof(int)));
  
  return 0;
}

//////////////////////////////////////////////////////////////////////
int NestedLoop::Run(int Nx, int *d_Ny, int i_func)
{
  return CumulSumNestedLoop(Nx, d_Ny, i_func);
}


//////////////////////////////////////////////////////////////////////
int NestedLoop::CumulSumNestedLoop(int Nx, int *d_Ny, int i_func)
{
  //TMP
  //double time_mark=getRealTime();
  //
  prefix_scan_.Scan(d_Ny_cumul_sum_, d_Ny, Nx+1);
  //TMP
  //printf("pst: %lf\n", getRealTime()-time_mark);
  //	 
  int Ny_sum;
  gpuErrchk(hipMemcpy(&Ny_sum, &d_Ny_cumul_sum_[Nx],
			  sizeof(int), hipMemcpyDeviceToHost));

  //printf("CSNL: %d %d\n", Nx, Ny_sum);
  
  //printf("Ny_sum %u\n", Ny_sum);
  //temporary - remove
  /*
  if (Ny_sum==0) {
    printf("Nx %d\n", Nx);
    for (int i=0; i<Nx+1; i++) {
      int psum;
      gpuErrchk(hipMemcpy(&psum, &d_Ny_cumul_sum_[i],
  			      sizeof(int), hipMemcpyDeviceToHost));
      printf("%d %d\n", i, psum);
    }
  }
  */    
  ////
  if(Ny_sum>0) {
    int grid_dim_x, grid_dim_y;
    if (Ny_sum<65536*1024) { // max grid dim * max block dim
      grid_dim_x = (Ny_sum+1023)/1024;
      grid_dim_y = 1;
    }
    else {
      grid_dim_x = 64; // I think it's not necessary to increase it
      if (Ny_sum>grid_dim_x*1024*65535l) {
	throw ngpu_exception(std::string("Ny sum ") + std::to_string(Ny_sum) +
			     " larger than threshold "
			     + std::to_string(grid_dim_x*1024*65535l));
      }
      grid_dim_y = (Ny_sum + grid_dim_x*1024 -1) / (grid_dim_x*1024);
    }
    dim3 numBlocks(grid_dim_x, grid_dim_y);
    //TMP
    //double time_mark=getRealTime();
    //
    switch (i_func) {
    case 0:
      CumulSumNestedLoopKernel0<<<numBlocks, 1024>>>
	(Nx, d_Ny_cumul_sum_, Ny_sum);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      break;
    case 1:
      CumulSumNestedLoopKernel1<<<numBlocks, 1024>>>
	(Nx, d_Ny_cumul_sum_, Ny_sum);
      gpuErrchk(hipPeekAtLastError());
      gpuErrchk(hipDeviceSynchronize());
      break;
    default:
      throw ngpu_exception("unknown nested loop function");
    }

    //TMP
    //printf("cst: %lf\n", getRealTime()-time_mark);
    //
  }
    
  return 0;
}

